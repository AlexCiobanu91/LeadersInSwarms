#include "hip/hip_runtime.h"
#include "Includes.cuh"
#include "Constants.cuh"
#include "ResearchKernel.cuh"
#include "ObstaclesKernel.cuh"
#include "SimpleAgents.cuh"
#include "LeadersKernel.cuh"

void sendData(std::string fis, Agent3DProperties* a3dpv, ObstacleProperties* opv, SimulationDetails* sd, 
			  GLuint agentVbo, GLuint obstaclesVbo, GLuint positionMatrixVbo)
{
	const int d3pv_size = sizeof(Agent3DProperties) * sd->swarmSize;
	const int sd_size = sizeof(SimulationDetails);
	const int rd_size = sizeof(ResearchDetails);
	const int d_osize = sizeof(ObstacleProperties) * sd->numberOfObstacles;
	const int f_size = sizeof(Frames);

	rd = new ResearchDetails();
	frames = new Frames();
	frames->frames = 0;

	hipMalloc((Frames**)&dd_frames, f_size);
	hipMalloc((Agent3DProperties**)&dd_d3pv, d3pv_size);
	hipMalloc((SimulationDetails**)&dd_sd, sd_size);
	hipMalloc((ResearchDetails**)&dd_rd, rd_size);
	hipMalloc((ObstacleProperties**)&dd_opv, d_osize);

	// copy host memory to device
	hipMemcpy(dd_frames, frames, f_size, hipMemcpyHostToDevice);
	hipMemcpy(dd_d3pv, a3dpv, d3pv_size, hipMemcpyHostToDevice);
	hipMemcpy(dd_opv, opv, d_osize, hipMemcpyHostToDevice);
	hipMemcpy(dd_sd, sd, sd_size, hipMemcpyHostToDevice);
	hipMemcpy(dd_rd, rd, rd_size, hipMemcpyHostToDevice);

	hipGraphicsGLRegisterBuffer(&obstacleResource, obstaclesVbo, cudaGraphicsMapFlagsNone);
	hipGraphicsGLRegisterBuffer(&agentsResource, agentVbo, cudaGraphicsMapFlagsNone);
  hipGraphicsGLRegisterBuffer(&positionMatrixResource, positionMatrixVbo, cudaGraphicsMapFlagsNone);

	InitFiles(fis);
}

 
void launchUpdateKernel(SimulationDetails* sd)
{
  size_t num_bytes;
  VertexFormat *dd_vfptr;
  //VertexFormat *dd_vfptr_pm;
  int num_threads = sd->swarmSize;
  int grid = glm::max((int)ceil((float)num_threads / MaxThreadsPerBlock), 1);

  hipGraphicsMapResources(1, &agentsResource, 0);
  hipGraphicsResourceGetMappedPointer((void **)&dd_vfptr, &num_bytes, agentsResource);

  updateKernel << <grid, MaxThreadsPerBlock >> > ((Agent3DProperties *)dd_d3pv, (VertexFormat *)dd_vfptr, (ObstacleProperties *)dd_opv, (SimulationDetails *)dd_sd);
  hipDeviceSynchronize();

  hipGraphicsUnmapResources(1, &agentsResource, 0);
}

void launchObstacleKernel(bool dynamicObstacles, SimulationDetails* sd) {
	VertexFormat *dd_ovfptr;
	size_t num_bytes;

	hipGraphicsMapResources(1, &obstacleResource, 0);
	hipGraphicsResourceGetMappedPointer((void **)&dd_ovfptr, &num_bytes, obstacleResource);

	launchObstacleKernel(sd, dynamicObstacles, dd_ovfptr);

	hipGraphicsUnmapResources(1, &obstacleResource, 0);
}

void launchRelocatePositionMatrixKernel(SimulationDetails* sd) 
{
  size_t num_bytes;
  VertexFormat *dd_vfptr_pm;
  int num_threads = sd->swarmSize;
  int grid = glm::max((int)ceil((float)num_threads / MaxThreadsPerBlock), 1);

  hipGraphicsMapResources(1, &positionMatrixResource, 0);
  hipGraphicsResourceGetMappedPointer((void **)&dd_vfptr_pm, &num_bytes, positionMatrixResource);

  relocatePositionMatrixKernel << <grid, MaxThreadsPerBlock >> > ((Agent3DProperties *)dd_d3pv, (VertexFormat *)dd_vfptr_pm,  (SimulationDetails *)dd_sd);
  hipDeviceSynchronize();

  hipGraphicsUnmapResources(1, &positionMatrixResource, 0);
}

extern "C" bool runMotivationLeader(const int argc, const char **argv, std::string fis, Agent3DProperties* a3dpv, ObstacleProperties* opv, SimulationDetails* sd, 
	bool dataSent, GLuint agentVbo, GLuint obstaclesVbo, GLuint positionMatrixVbo, bool dynamicObstacles, bool freeMemory = false) {

	if (freeMemory) {

    hipFree(dd_frames);
    hipFree(dd_d3pv);
    hipFree(dd_sd);
    hipFree(dd_rd);
    hipFree(dd_opv);

		closeFiles();
		hipDeviceReset();
	}
	
	if (sd != NULL && a3dpv != NULL && opv != NULL) {
		if (!dataSent) {
			sendData(fis, a3dpv, opv, sd, agentVbo, obstaclesVbo, positionMatrixVbo);
    }
    else {
      const int sd_size = sizeof(SimulationDetails);
      hipMemcpy(dd_sd, sd, sd_size, hipMemcpyHostToDevice);
    }

		launchUpdateKernel(sd);
    // launchRelocatePositionMatrixKernel(sd);
		// launchObstacleKernel(dynamicObstacles, sd);
    hipMemcpy(sd, dd_sd, sizeof(SimulationDetails), hipMemcpyDeviceToHost);

		// launchResearchKernel(sd);		

    
	}
	
	return true;
}
